
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world(void) {
    printf("GPU: Hello World!\n");
}


int main(void) {

    printf("CPU: Hello World!\n");

    hello_world<<<1, 10>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceReset();

    return 0;

}