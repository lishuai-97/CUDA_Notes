
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int b = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    printf("Hello World form block-%d and thread-(%d, %d)!\n", b, tx, ty);
}


int main(void)
{
    const dim3 block_size(2, 4);
    hello_from_gpu<<<1, block_size>>>();
    hipDeviceSynchronize();
    return 0;
}