#include "hip/hip_runtime.h"
#include "foo.cuh"
#include <hip/hip_runtime.h>
#include <>
 
//Kernel
__global__ void d_vec_add(int *d_a, int *d_b, int *d_c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        d_c[i] = d_a[i] + d_b[i];
}
 
void h_vec_add(int *a, int *b, int *c, int n)
{
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, sizeof(int) * n);
    hipMalloc((void **)&d_b, sizeof(int) * n);
    hipMalloc((void **)&d_c, sizeof(int) * n);
 
    hipMemcpy(d_a, a, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * n, hipMemcpyHostToDevice);
 
    dim3 DimGrid(n / BX + 1, 1, 1);
    dim3 DimBlock(BX, 1, 1);
 
    d_vec_add<<<DimGrid, DimBlock>>>(d_a, d_b, d_c, n);
 
    hipMemcpy(c, d_c, sizeof(int) * n, hipMemcpyDeviceToHost);
 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
 