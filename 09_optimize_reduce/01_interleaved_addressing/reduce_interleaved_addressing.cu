#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int len = 32 * 1024 * 1024;

template <int BLOCKSIZE>
__global__ void reduce_naive_kernel(int *arr, int *out, int len)
{
    __shared__ int sdata[BLOCKSIZE];
    int tid = threadIdx.x;    // 线程 id (block 内)
    int bid = blockIdx.x;     // block id (grid 内)
    int bdim = blockDim.x;    // block 大小
    int i = bid * bdim + tid; // 全局 id

    // 将数据拷贝到共享内存
    if (i < len)
    {
        sdata[tid] = arr[i];
    }

    __syncthreads(); // 等待所有线程完成

    // 使用交错寻址
    for (int s = 1; s < bdim; s *= 2)
    {   
        // 计算当前线程应该处理的元素索引
        // 每轮迭代中，活跃线程数减半，但每个线程处理的元素位置改变
        int index = 2 * s * tid;
        // index + s < bdim 确保在共享内存范围内
        // bdim * bid + s < len 确保在全局内存范围内
        if ((index + s < bdim) && (bdim * bid + s < len))
        {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    // 每个 block 的第一个线程将结果写入到 out 中
    if (tid == 0)
    {
        out[bid] = sdata[0];
    }
}

int main()
{
    int *arr = new int[len];
    int *out = new int[len];
    int *d_arr, *d_out;

    // 初始化数组
    for (int i = 0; i < len; i++)
    {
        arr[i] = 1;
    }

    // 分配内存
    hipMalloc((void **)&d_arr, sizeof(int) * len);
    hipMalloc((void **)&d_out, sizeof(int) * len);

    // 拷贝数据到显存
    hipMemcpy(d_arr, arr, sizeof(int) * len, hipMemcpyHostToDevice);

    // 计算 block 和 grid 的大小
    const int blocksize = 256;
    const int gridsize = (len + blocksize - 1) / blocksize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipEventQuery(start);

    // 调用 kernel 函数
    reduce_naive_kernel<blocksize><<<gridsize, blocksize>>>(d_arr, d_out, len);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("elapsed time: %f ms.\n", elapsed_time);
    printf("elapsed time: %f us.\n", elapsed_time * 1000);

    // 拷贝数据到内存
    hipMemcpy(out, d_out, sizeof(int) * len, hipMemcpyDeviceToHost);

    // 计算结果
    long long sum = 0;
    for (int i = 0; i < gridsize; i++)
    {
        sum += out[i];
    }
    printf("sum = %d\n", sum);

    // 核对结果
    long long sum2 = 0;
    for (int i = 0; i < len; i++)
    {
        sum2 += arr[i];
    }

    if (sum == sum2)
    {
        printf("success\n");
    }
    else
    {
        printf("failed, the result is %d\n", sum2);
    }

    // 释放内存
    hipFree(d_arr);
    hipFree(d_out);
    delete[] arr;
    delete[] out;
    return 0;
}