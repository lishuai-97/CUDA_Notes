#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int len = 10000;

template <int BLOCKSIZE>
__global__ void reduce_naive_kernel(int *arr, int *out, int len)
{
    __shared__ int sdata[BLOCKSIZE];
    int tid = threadIdx.x;    // 线程 id (block 内)
    int bid = blockIdx.x;     // block id (grid 内)
    int bdim = blockDim.x;    // block 大小
    int i = bid * bdim + tid; // 全局 id

    // 将数据拷贝到共享内存
    if (i < len)
    {
        sdata[tid] = arr[i];
    }

    __syncthreads(); // 等待所有线程完成

    // 每个线程计算 bdim^0.5 个轮回
    // 比如 bdim = 8, 则每个线程计算 2 个轮回
    for (int s = 1; s < bdim; s *= 2)
    {
        if (tid % (2 * s) == 0 && i + s < len)
        {
            sdata[tid] += sdata[tid + s];
        }
        // 等待所有线程完成 后再进行下一轮计算
        __syncthreads();
    }

    // 每个 block 的第一个线程将结果写入到 out 中
    if (tid == 0)
    {
        out[bid] = sdata[0];
    }
}

int main()
{
    int *arr = new int[len];
    int *out = new int[len];
    int *d_arr, *d_out;

    // 初始化数组
    for (int i = 0; i < len; i++)
    {
        arr[i] = i;
    }

    // 分配内存
    hipMalloc((void **)&d_arr, sizeof(int) * len);
    hipMalloc((void **)&d_out, sizeof(int) * len);

    // 拷贝数据到显存
    hipMemcpy(d_arr, arr, sizeof(int) * len, hipMemcpyHostToDevice);

    // 计算 block 和 grid 的大小
    const int blocksize = 32;
    const int gridsize = (len + blocksize - 1) / blocksize;

    // 调用 kernel 函数
    reduce_naive_kernel<blocksize><<<gridsize, blocksize>>>(d_arr, d_out, len);

    // 拷贝数据到内存
    hipMemcpy(out, d_out, sizeof(int) * len, hipMemcpyDeviceToHost);

    // 计算结果
    int sum = 0;
    for (int i = 0; i < gridsize; i++)
    {
        sum += out[i];
    }
    printf("sum = %d\n", sum);

    // 核对结果
    int sum2 = 0;
    for (int i = 0; i < len; i++)
    {
        sum2 += arr[i];
    }

    if (sum == sum2)
    {
        printf("success\n");
    }
    else
    {
        printf("failed\n");
    }

    // 释放内存
    hipFree(d_arr);
    hipFree(d_out);
    delete[] arr;
    delete[] out;
    return 0;
}